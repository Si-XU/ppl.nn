#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include <fstream>
#include <iostream>
using namespace std;

#include <vector>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <fstream>
#include <sstream>
#include <string>
#include <unordered_map>

#include "cudakernel/nn/conv/conv_fp16.h"
#include "kernel_type.h"
#include "conv_common.h"
#include "common/init_lut.h"
#include "common/merge_split.h"

#include "float.h"

#define TIMES 4

#define SPK_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                                                                  \
    		in_lut,                        in_lut_size,                                 \
    		flt_lut,                       flt_lut_size,                                \
            num_chl_per_spk_head,          num_chl_per_spk_tail,                        \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           (int *)bias

#define LUT_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                                                                  \
    		in_lut,                        in_lut_size,                                 \
    		flt_lut,                       flt_lut_size,                                \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

#define SWZL_SPK_KPARAM_LIST \
            d_flt,                                                                      \
            pad_input,                                                                  \
            conv_out,                                                                   \
            kloop_num,                                                                  \
            in_lut,                        in_lut_size,                                 \
            flt_lut,                       flt_lut_size,                                \
            num_chl_per_spk_head,          num_chl_per_spk_tail,                        \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           (int *)bias

#define SWZL_LUT_KPARAM_LIST \
            d_flt,                                                                      \
            pad_input,                                                                  \
            conv_out,                                                                   \
            kloop_num,                                                                  \
            in_lut,                        in_lut_size,                                 \
            flt_lut,                       flt_lut_size,                                \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

#define IDX_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                      koff_num_pad,                               \
            in_hw,                         out_hw,                                      \
            flt_hw,                        out_nhw,                                     \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            conv_param.num_chl,            num_chl_per_grp,                             \
            in_chl_per_grp_pad,            flt_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

#define MERGE_KPARAM_LIST \
        	conv_out,                      final_out,                                   \
        	spk_height_v1,                 spk_width_v8,                                \
        	out_hw,                        splitk * splitf,                             \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

static std::vector<kernel_info_t> g_kernel_container;
static bool is_g_kernel_container_initialized = false;

static std::unordered_map<size_t, algo_param_t> g_conv_shape_hash;

void InitializeKernelContainer(std::vector<kernel_info_t> &g_kernel_container, ppl::common::datatype_t type)
{
    if( type == ppl::common::DATATYPE_FLOAT16 ) {
        Initialize2spkConvF1KernelContainer(g_kernel_container);
        Initialize2spkConvF3KernelContainer(g_kernel_container);
        Initialize2spkConvFNKernelContainer(g_kernel_container);
        Initialize2spkConvFSKernelContainer(g_kernel_container);
                      
        InitializeIdxnConvKernelContainer(g_kernel_container);

        InitializeSwzlConvF1KernelContainer(g_kernel_container);
        InitializeSwzlConvF3KernelContainer(g_kernel_container);
        InitializeSwzlConvFNKernelContainer(g_kernel_container);
    }
    
    is_g_kernel_container_initialized = true;
}

__inline__ std::string GetConvShapeString(conv_param_t &conv_param)
{
    return std::string("b" + std::to_string(conv_param.in_num)  + \
                       "_c" + std::to_string(conv_param.num_chl) + \
                       "_d" + std::to_string(conv_param.num_flt) + \
                       "_g" + std::to_string(conv_param.num_grp) + \
                       "_h" + std::to_string(conv_param.in_height) + \
                       "_w" + std::to_string(conv_param.in_width) + \
                       "_r" + std::to_string(conv_param.flt_height) + \
                       "_s" + std::to_string(conv_param.flt_width) + \
                       "_p" + std::to_string(conv_param.pad_height) + \
                       "_q" + std::to_string(conv_param.pad_width) + \
                       "_u" + std::to_string(conv_param.stride_height) + \
                       "_v" + std::to_string(conv_param.stride_width) + \
                       "_y" + std::to_string(conv_param.hole_height) + \
                       "_x" + std::to_string(conv_param.hole_width) + \
                       "_");
}

__inline__ size_t GetConvShapeHashKey( conv_param_t &conv_param )
{
    return std::hash<std::string>{} (GetConvShapeString(conv_param));
}

uint64_t PPLCUDAConvolutionGetCompilationBufSize(ppl::common::datatype_t type, conv_param_t &conv_param, uint64_t workspace)
{
    int pad_size = GetPadSize(type);

    uint32_t num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    uint32_t num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    uint32_t num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    uint32_t num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size); 

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint32_t cvt_input_size = 0;
    uint32_t cvt_output_size = 0;

    if(is_in_grp_pad)
        cvt_input_size = GetCvtInputSize( type, conv_param, num_chl_per_grp_pad);

    if(is_out_grp_pad)
        cvt_output_size = getCvtOutputSize(type, conv_param, num_flt_per_grp_pad);

    uint32_t split_size = GetMaxSplitSize(type, conv_param, num_flt_per_grp_pad);

    uint64_t total_size = cvt_input_size + cvt_output_size + split_size;

    return total_size <= workspace ? total_size : workspace;
}
uint64_t PPLCUDAConvolutionGetRuntimeBufSize(
        ppl::common::datatype_t type,
        conv_param_t &conv_param,
        unsigned int splitk,
        unsigned int splitf,
        uint64_t workspace)
{
    int pad_size = GetPadSize(type);

    uint32_t num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    uint32_t num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    uint32_t num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    uint32_t num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size); 

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint32_t cvt_input_size = 0;
    uint32_t cvt_output_size = 0;

    if(is_in_grp_pad)
        cvt_input_size = GetCvtInputSize(type, conv_param, num_chl_per_grp_pad);
    if(is_out_grp_pad)
        cvt_output_size = getCvtOutputSize(type, conv_param, num_flt_per_grp_pad);

    uint32_t split_size = 0;
    
    if(splitk > 1 || splitf > 1)
        split_size = GetSplitKFSize(type, conv_param, num_flt_per_grp_pad, splitk, splitf);

    uint64_t total_size  = cvt_input_size + cvt_output_size + split_size;

    return total_size <= workspace ? total_size : workspace;
}

ppl::common::RetCode PPLCUDAConvolutionQuickSelectKernel(
        select_param_t &select_param,
        conv_param_t &conv_param) {
    int in_hw = conv_param.in_num * conv_param.in_height * conv_param.in_width;
    int out_hw = conv_param.in_num * conv_param.out_height * conv_param.out_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int chl_per_group = conv_param.num_chl / conv_param.num_grp;

    // Use non-shared memory algo for small channel
    if (chl_per_group < 64) {
        if (flt_hw > 9) {
            select_param.m_cta = 128;
            select_param.m_warp = 64;
        } else {
            select_param.m_cta = 32;
            select_param.m_warp = 16;
        }

        if (in_hw == out_hw) {
            select_param.n_cta = 64;
            select_param.n_warp = 32;
        } else {
            select_param.n_cta = 32;
            select_param.n_warp = 16;
        }

        if (conv_param.num_chl >= 16) {
            select_param.k_cta = 32;
            select_param.k_warp = 32;
        } else {
            select_param.k_cta = 16;
            select_param.k_warp = 16;
        }
    } else { // Use 3spk algo for large channel
        float min_pad = 1.0;
        select_param.m_cta = 16;
        for (int32_t i = 128; i >= 16; i = i / 2) {
            if (out_hw < i) continue;
            float pad = 1.0 * (DivUp(out_hw, i) * i - out_hw) / out_hw;
            if (pad < min_pad)  {
                min_pad = pad;
                select_param.m_cta = i;
            }
            if (min_pad < 0.1)  break;
        }

        select_param.n_cta = 16;
        for (int32_t i = 128; i >= 16; i = i / 2) {
            int cout = conv_param.num_flt;
            if ((cout < 64 && i / cout == 1) || (cout >= 64 && cout % i == 0)) {
                select_param.n_cta = i;
                break;
            }
        }

        if (conv_param.num_chl >= 128) {
            select_param.k_cta = 64;
        } else {
            select_param.k_cta = 32;
        }

        if (select_param.m_cta == 128 && select_param.n_cta == 128) {
            select_param.m_cta = 64;
        }

        if (select_param.m_cta * 4 < select_param.n_cta) {
            select_param.m_cta *= 2;
            select_param.n_cta /= 2;
        }
        if (select_param.n_cta *4 < select_param.m_cta) {
            select_param.m_cta /= 2;
            select_param.n_cta *= 2;
        }

        select_param.m_warp = select_param.m_cta / 2;
        select_param.n_warp = select_param.n_cta / 2;
        select_param.k_warp = select_param.k_cta / 2;
        if (select_param.k_warp < 8) {
            select_param.k_warp = 16;
        }
    }
    select_param.quick_select = true;
    return ppl::common::RC_SUCCESS;
}

ppl::common::RetCode PPLCUDAConvolutionSelectKernel(
        hipStream_t &stream, 
        ppl::common::datatype_t type,
        int4* d_input,
        int4* d_flt,
        int4* d_output,
        int4* bias,
        int4* d_temp_buf, 
        algo_param_t & algo_param,
        conv_param_t &conv_param, 
        fuse_param_t &fuse_param,
        select_param_t &select_param,
        std::string node_name,
	    uint64_t workspace)
{

    int selected_count = 0;
    // ofstream outfile;
    // outfile.open(node_name + ".csv");

    if(!is_g_kernel_container_initialized)
        InitializeKernelContainer(g_kernel_container, type);

    size_t conv_shape_hash = GetConvShapeHashKey(conv_param);

    std::unordered_map<size_t, algo_param_t>::const_iterator conv_shape_hash_iterator = g_conv_shape_hash.find(conv_shape_hash);

    if(conv_shape_hash_iterator != g_conv_shape_hash.end()) {
        algo_param.kid    = conv_shape_hash_iterator->second.kid;
        algo_param.splitk = conv_shape_hash_iterator->second.splitk;
        algo_param.splitf = conv_shape_hash_iterator->second.splitf;

        return ppl::common::RC_SUCCESS;
    }

    int pad_size = GetPadSize(type);

    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    int num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    int num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size);

    int in_hw = conv_param.in_height * conv_param.in_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int out_hw = conv_param.out_height * conv_param.out_width;

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint64_t buf_off_v4 = 0;

    int4 *pad_input = d_input;
    int4 *pad_output = d_output;

    if(is_in_grp_pad) {
	    pad_input = d_temp_buf; 
	    buf_off_v4 += GetCvtInputSize(type, conv_param, num_chl_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);

        PPLCUDAConvolutionCvtInput(stream, pad_input, d_input, type, conv_param);
    }

    if(is_out_grp_pad) {
	    pad_output = d_temp_buf + buf_off_v4;
	    buf_off_v4 += getCvtOutputSize(type, conv_param, num_flt_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);
    } 

    int4 * final_out = fuse_param.has_concat ? (int4 *) fuse_param.post_concat : pad_output;

    int4 *splitk_buf = d_temp_buf + buf_off_v4;

    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    float minTime = FLT_MAX;

    float elapsed;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    const int SPLITK_OPTIONS[] = {1, 2, 4, 8};
    for(unsigned int spk = 0; spk < 4; spk++) {
        unsigned int splitk = SPLITK_OPTIONS[spk];

        for(unsigned int kid = 0; kid < g_kernel_container.size(); kid++) {
            unsigned int splitf = (g_kernel_container[kid].ktype == CONV_2SPK_FS) ? flt_hw : 1;
        
            if(!g_kernel_container[kid].CheckKernelTypeFeasible(conv_param.flt_height, conv_param.flt_width, num_chl_per_grp, splitk)) continue;

            if(!g_kernel_container[kid].CheckSplitkFeasible(num_chl_per_grp, splitk)) continue;

            if(!g_kernel_container[kid].CheckSplitfFeasible(splitf, splitk)) continue;

            if(!g_kernel_container[kid].CheckQuickSelectFeasible(select_param, conv_param.num_chl / conv_param.num_grp, splitk, splitf)) continue;

            int4 *conv_out = (splitk > 1 || splitf > 1) ? splitk_buf : final_out;

            dim3 block_size, grid_size;

            block_size.x = g_kernel_container[kid].cta_size_in_thd;
            block_size.y = 1;
            block_size.z = 1;

            if(g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
                    g_kernel_container[kid].ktype == CONV_SWZL_FN) {
                grid_size.x = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_n_per_cta);
                grid_size.y = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_m_per_cta);
            } else {
                grid_size.x = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_m_per_cta);
                grid_size.y = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_n_per_cta);
            }

            grid_size.z = conv_param.num_grp * splitk * splitf;

            if(!g_kernel_container[kid].CheckSplitkBlockSizeFeasible((grid_size.x * grid_size.y * grid_size.z), splitk)) continue;

	        hipEventRecord(begin, stream);

	        for(int i = 0; i < TIMES; i++) {
                if(g_kernel_container[kid].ktype == CONV_IDXN_C2 || g_kernel_container[kid].ktype == CONV_IDXN_C4 || \
                        g_kernel_container[kid].ktype == CONV_IDXN_C32) {
                    int tile_k_per_step = g_kernel_container[kid].tile_k_per_step;

                    int img_pad_size    = pad_size;
                    int flt_pad_size    = g_kernel_container[kid].flt_pad_size;
                    int out_nhw         = out_hw * conv_param.in_num;

                    int in_chl_per_grp_pad = Align(num_chl_per_grp, img_pad_size);
                    int flt_chl_per_grp_pad = Align(num_chl_per_grp, flt_pad_size);
                    int num_flt_per_grp_pad = Align(num_flt_per_grp, img_pad_size);

	                int kloop_num        = DivUp(flt_hw * flt_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);
                    int koff_num_pad      = Align(kloop_num * (g_kernel_container[kid].tile_k_per_cta / flt_pad_size), WARP_SIZE);

                    (g_kernel_container[kid].idx_kptr)<<<grid_size, block_size, 0, stream>>>(IDX_KPARAM_LIST);
                }
                else if(g_kernel_container[kid].ktype == CONV_2SPK_F1 || g_kernel_container[kid].ktype == CONV_2SPK_F3 || \
                        g_kernel_container[kid].ktype == CONV_2SPK_FN || g_kernel_container[kid].ktype == CONV_2SPK_FS || \
                        g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
                        g_kernel_container[kid].ktype == CONV_SWZL_FN) {

	                int kloop_num = (flt_hw / splitf) * DivUp(num_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);

                    lut_t in_lut, flt_lut;
                    int in_lut_size, flt_lut_size;
                
                    InitializeInputLut(in_lut_size, in_lut.idx, conv_param.flt_height, conv_param.flt_width, conv_param.in_height,
                            conv_param.in_width, conv_param.pad_height, conv_param.pad_width, conv_param.hole_height, conv_param.hole_width,
                            num_chl_per_grp_pad, conv_param.num_grp, g_kernel_container[kid].tile_k_per_cta, pad_size);

                    InitializeFilterLut(flt_lut_size, flt_lut.idx, conv_param.flt_height, conv_param.flt_width, num_chl_per_grp_pad,
                            g_kernel_container[kid].tile_k_per_cta, pad_size);

                    if(splitk == 1) {
                        if(g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
                                g_kernel_container[kid].ktype == CONV_SWZL_FN)
                            (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(SWZL_LUT_KPARAM_LIST);
                        else
                            (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(LUT_KPARAM_LIST);
                    } else {
                        int num_chl_per_spk_head, num_chl_per_spk_tail;

                        InitializeNumChlPerSpk(num_chl_per_spk_head, num_chl_per_spk_tail, conv_param.num_chl, conv_param.num_grp, 
                                pad_size, g_kernel_container[kid].tile_k_per_cta, splitk);

                        if(g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
                                g_kernel_container[kid].ktype == CONV_SWZL_FN)
                            (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SWZL_SPK_KPARAM_LIST);
                        else
                            (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SPK_KPARAM_LIST);
                    }

                    if(splitk > 1 || splitf > 1) {
                        int spk_width_v8   = num_flt_per_grp_pad * conv_param.num_grp / pad_size;
                        int spk_height_v1  = out_hw * conv_param.in_num;

                        dim3 merge_grid_size, merge_block_size;
                        merge_block_size.x = 64; // empirical value
                        merge_block_size.y = 1;
                        merge_block_size.z = 1;

                        merge_grid_size.x  = spk_height_v1;
                        merge_grid_size.y  = DivUp(spk_width_v8, merge_block_size.x);
                        merge_grid_size.z  = 1;

                        MergeConvSplitResults<<<merge_grid_size, merge_block_size, 0, stream>>>(MERGE_KPARAM_LIST);
                    }
                }
            }

	        hipEventRecord(end, stream);
	        hipEventSynchronize(end);
	        hipEventElapsedTime(&elapsed, begin, end);

	        if(elapsed < minTime){
                algo_param.kid = kid;
                algo_param.splitk = splitk;
                algo_param.splitf = splitf;
	            minTime = elapsed;
	        }

            // outfile << kid << "," << elapsed << ","
            //     << in_hw << ","
            //     << flt_hw << ","
            //     << out_hw << ","
            //     << conv_param.num_flt << ","
            //     << conv_param.num_chl << ","
            //     << g_kernel_container[kid].tile_m_per_cta << ","
            //     << g_kernel_container[kid].tile_n_per_cta << ","
            //     << g_kernel_container[kid].tile_k_per_cta << ","
            //     << g_kernel_container[kid].tile_m_per_warp << ","
            //     << g_kernel_container[kid].tile_n_per_warp << ","
            //     << g_kernel_container[kid].tile_k_per_warp << ","
            //     << g_kernel_container[kid].tile_k_per_step << ","
            //     << g_kernel_container[kid].tile_k_per_set << ","
            //     << g_kernel_container[kid].flt_size << ","
            //     << g_kernel_container[kid].flt_pad_size << ","
            //     << g_kernel_container[kid].cta_size_in_thd << ","
            //     << g_kernel_container[kid].kname << ","
            //     << splitk << "," << splitf << endl;
            if (splitk == 1)   selected_count++;
        }
    }

    if(is_out_grp_pad) {
        PPLCUDAConvolutionCvtOutput(stream, d_output, final_out, type, conv_param);
    }

    hipEventDestroy(begin);
    hipEventDestroy(end);

    g_conv_shape_hash[conv_shape_hash] = algo_param;
    printf("%s,%d,%d,%d\n", GetConvShapeString(conv_param).data(), algo_param.kid, algo_param.splitk, algo_param.splitf);
    // outfile.close();

    return ppl::common::RC_SUCCESS;
}

void PPLCUDAConvolutionForwardImp(
        hipStream_t &stream, 
        ppl::common::datatype_t type,
        int4* d_input,
        int4* d_flt,
        int4* d_output,
        int4* bias,
        int4* d_temp_buf,
        algo_param_t& algo_param,
        conv_param_t &conv_param,
        fuse_param_t &fuse_param)
{
    if(!is_g_kernel_container_initialized)
        InitializeKernelContainer(g_kernel_container, type);

    unsigned int kid = algo_param.kid;
    unsigned int splitk = algo_param.splitk;
    unsigned int splitf = algo_param.splitf;

    int pad_size = GetPadSize(type);

    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    int num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    int num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size);

    int in_hw  = conv_param.in_height * conv_param.in_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int out_hw = conv_param.out_height * conv_param.out_width;

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint64_t buf_off_v4 = 0;

    int4 *pad_input = d_input;
    int4 *pad_output = d_output;

    if(is_in_grp_pad) {
	    pad_input = d_temp_buf; 
	    buf_off_v4 += GetCvtInputSize(type, conv_param, num_chl_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);

        PPLCUDAConvolutionCvtInput(stream, pad_input, d_input, type, conv_param);
    }

    if(is_out_grp_pad) {
	    pad_output = d_temp_buf + buf_off_v4;
	    buf_off_v4 += getCvtOutputSize(type, conv_param, num_flt_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);
    } 

    int4 *final_out  = fuse_param.has_concat ? (int4 *) fuse_param.post_concat : pad_output;

    int4 *splitk_buf = d_temp_buf + buf_off_v4;
    int4 *conv_out   = (splitk > 1 || splitf > 1) ? splitk_buf : final_out;

    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    dim3 block_size, grid_size;

    block_size.x = g_kernel_container[kid].cta_size_in_thd;
    block_size.y = 1;
    block_size.z = 1;

    if(g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
            g_kernel_container[kid].ktype == CONV_SWZL_FN) {
        grid_size.x = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_n_per_cta);
        grid_size.y = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_m_per_cta);
    } else {
        grid_size.x = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_m_per_cta);
        grid_size.y = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_n_per_cta);
    }

    grid_size.z = conv_param.num_grp * splitk * splitf;

    if(g_kernel_container[kid].ktype == CONV_IDXN_C2 || g_kernel_container[kid].ktype == CONV_IDXN_C4 || \
            g_kernel_container[kid].ktype == CONV_IDXN_C32) {
        int img_pad_size = pad_size;
        int flt_pad_size = g_kernel_container[kid].flt_pad_size;

        int out_nhw = out_hw * conv_param.in_num;

        int in_chl_per_grp_pad = Align(num_chl_per_grp, img_pad_size);
        int flt_chl_per_grp_pad = Align(num_chl_per_grp, flt_pad_size);
        int num_flt_per_grp_pad = Align(num_flt_per_grp, img_pad_size);

	    int kloop_num = DivUp(flt_hw * flt_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);
        int koff_num_pad = Align(kloop_num * (g_kernel_container[kid].tile_k_per_cta / flt_pad_size), WARP_SIZE);

        (g_kernel_container[kid].idx_kptr)<<<grid_size, block_size, 0, stream>>>(IDX_KPARAM_LIST);

    } else if(g_kernel_container[kid].ktype == CONV_2SPK_F1 || g_kernel_container[kid].ktype == CONV_2SPK_F3 || \
              g_kernel_container[kid].ktype == CONV_2SPK_FN || g_kernel_container[kid].ktype == CONV_2SPK_FS || \
              g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
              g_kernel_container[kid].ktype == CONV_SWZL_FN) {

	    int kloop_num = (flt_hw / splitf) * DivUp(num_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);

        lut_t in_lut, flt_lut;
        int in_lut_size, flt_lut_size;
    
        InitializeInputLut(in_lut_size, in_lut.idx, conv_param.flt_height, conv_param.flt_width, conv_param.in_height,
                conv_param.in_width, conv_param.pad_height, conv_param.pad_width, conv_param.hole_height, conv_param.hole_width,
                num_chl_per_grp_pad, conv_param.num_grp, g_kernel_container[kid].tile_k_per_cta, pad_size);

        InitializeFilterLut(flt_lut_size, flt_lut.idx, conv_param.flt_height, conv_param.flt_width, num_chl_per_grp_pad,
                g_kernel_container[kid].tile_k_per_cta, pad_size);

        if(splitk == 1) {
            if(g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
                    g_kernel_container[kid].ktype == CONV_SWZL_FN)
                (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(SWZL_LUT_KPARAM_LIST);
            else
                (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(LUT_KPARAM_LIST);
        } else {
            int num_chl_per_spk_head, num_chl_per_spk_tail;

            InitializeNumChlPerSpk(num_chl_per_spk_head, num_chl_per_spk_tail, conv_param.num_chl, conv_param.num_grp, 
                    pad_size, g_kernel_container[kid].tile_k_per_cta, splitk);

            if(g_kernel_container[kid].ktype == CONV_SWZL_F1 || g_kernel_container[kid].ktype == CONV_SWZL_F3 || \
                    g_kernel_container[kid].ktype == CONV_SWZL_FN)
                (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SWZL_SPK_KPARAM_LIST);
            else
                (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SPK_KPARAM_LIST);
        }
    }
    
    if(splitk > 1 || splitf > 1) {
        int spk_width_v8   = num_flt_per_grp_pad * conv_param.num_grp / pad_size;
        int spk_height_v1  = out_hw * conv_param.in_num;

        dim3 merge_grid_size, merge_block_size;
        merge_block_size.x = 64;
        merge_block_size.y = 1;
        merge_block_size.z = 1;

        merge_grid_size.x  = spk_height_v1;
        merge_grid_size.y  = DivUp(spk_width_v8, merge_block_size.x);
        merge_grid_size.z  = 1;

        MergeConvSplitResults<<<merge_grid_size, merge_block_size, 0, stream>>>(MERGE_KPARAM_LIST);
    }

    if(is_out_grp_pad) {
        PPLCUDAConvolutionCvtOutput(stream, d_output, final_out, type, conv_param);
    }
    
}
