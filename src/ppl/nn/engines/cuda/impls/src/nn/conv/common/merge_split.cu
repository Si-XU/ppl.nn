// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#define _4HALF2_            4
#define _8HALF_             8
#define _INT4_TO_4INT_      4
#define _INT4_TO_8HALF_     8

#define HMAX2_INST(_d, _a, _b, _c) \
        asm volatile("vmax2.s32.s32.s32 %0, %1, %2, %3;\n":   "=r"(_d): "r"(_a), "r"(_b), "r"(_c));

#define HMIN2_INST(_d, _a, _b, _c) \
        asm volatile("vmin2.s32.s32.s32 %0, %1, %2, %3;\n":   "=r"(_d): "r"(_a), "r"(_b), "r"(_c));


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void MergeConvSplitResults(
        int4* input,             int4* output, 
	    int split_height_v1,     int split_width_v8, 
	    int out_hw,              int split, 
        int has_bias,            const int4* bias,
        int  has_relu,           const int clip_min,
        bool has_clip,           const int clip_max,
        bool has_elt,            const int4* pre_data,
        int  has_elt_relu,       const int elt_clip_min,
        bool has_elt_clip,       const int elt_clip_max,
        bool has_concat,         int concat_offset_v8,
        int concat_stride_v8)
{
#if (__CUDA_ARCH__ >= 600) && (__CUDACC_VER_MAJOR__ >= 9)
    int k_id = blockIdx.y * blockDim.x + threadIdx.x;
    int64_t nhw_id = blockIdx.x;

    int off  = nhw_id * split_width_v8 + k_id;

    const int4 ZEROv4 = {0, 0, 0, 0};
    bool is_in_range = k_id < split_width_v8;

    int4 merge_v4, split_v4, bias_v4;

    __half2 * h2_merge = (__half2 *) &merge_v4;
    __half2 * h2_split = (__half2 *) &split_v4;
    __half2 * h2_bias  = (__half2 *) &bias_v4;

    merge_v4 = is_in_range ? input[off] : ZEROv4;

    for(int i = 1; i < split; i++)
    {
        split_v4 = is_in_range ? input[off + i * split_height_v1 * split_width_v8] : ZEROv4;

	    for(int j = 0; j < _4HALF2_; j++)
	        h2_merge[j] = __hadd2(h2_merge[j], h2_split[j]);
    }
    if(has_bias)
    {
        bias_v4 = is_in_range ? ((int4 *) bias) [k_id] : ZEROv4;

#pragma unroll
	    for(int j = 0; j < _4HALF2_; j++)
	        h2_merge[j] = __hadd2(h2_merge[j], h2_bias[j]);
    }

    int *    merge_v1  = (int *)    &merge_v4;

    if(has_relu)
    {
        for(int i = 0; i < _4HALF2_; i++)
            merge_v1[i] = __vmaxs2(merge_v1[i], 0);
    }

    if(has_clip) {
#pragma unroll
        for(int i = 0; i < _4HALF2_; i++)
        {
            HMIN2_INST(merge_v1[i], merge_v1[i], clip_max, merge_v1[i]); \
            HMAX2_INST(merge_v1[i], merge_v1[i], clip_min, merge_v1[i]); \
        }
    }

    if(has_elt) {
	    int4 eltV4     = is_in_range ? pre_data[off] : ZEROv4;
	    __half2* h2Elt = (__half2*) &eltV4;

	    for(int i = 0; i < _INT4_TO_4INT_; i++)
	        h2_merge[i] = __hadd2(h2_merge[i], h2Elt[i]);
    }

    if(has_elt_relu) {
        for(int i = 0; i < _4HALF2_; i++)
            merge_v1[i] = __vmaxs2(merge_v1[i], 0);
    }

    if(has_elt_clip) {
        for(int i = 0; i < _4HALF2_; i++) {
            HMIN2_INST(merge_v1[i], merge_v1[i], elt_clip_max, merge_v1[i]); \
            HMAX2_INST(merge_v1[i], merge_v1[i], elt_clip_min, merge_v1[i]); \
        }
    }

    int concat_v8_off = 0;
    if(has_concat){
	    concat_v8_off = concat_offset_v8 + nhw_id * concat_stride_v8;
	    off = concat_v8_off + k_id;
    }
    
    if(is_in_range) output[off] = merge_v4;

#endif
}
